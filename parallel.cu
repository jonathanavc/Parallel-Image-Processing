#include "hip/hip_runtime.h"
// nvcc parallel_linear_interpolation.cu -lX11 -o parallel_BLI


#include <iostream>
#include <dirent.h>
#include <hip/hip_runtime.h>
#include "CImg.h"
#include "./others/metrictime.hpp"

using namespace cimg_library;
using namespace std;

static int block_size = 1024;

static int pixel_per_thread = 8;

__device__ int pixel(unsigned char *img, int x, int y, int width, int size, int rgb){
    return img[(x) + (y)*width + size * rgb];
}

__global__ void linear_interpolation(unsigned char *d_old_image, unsigned char *d_new_image, int old_width, int old_height, int new_width, int new_height,int  scale,int pixel_per_thread){
    int old_size = old_height * old_width;
    int new_size = new_height * new_width;
    for (int i = 0; i < pixel_per_thread; i++){
        int pos = pixel_per_thread *(blockIdx.x * blockDim.x + threadIdx.x) + i;
        int pos_x = (pos % new_size) % new_width;
        int pos_y = (pos % new_size) / new_width;
        int r_g_b = pos / new_size;
        if (pos_x >= new_width || pos_y >= new_height) continue;
        if (pos_x % scale == 0 && pos_y % scale == 0) d_new_image[pos_x + pos_y * new_width + new_size * r_g_b] = pixel(d_old_image, pos_x / scale, pos_y / scale, old_width, old_size, r_g_b);
        else if (pos_x % scale == 0) d_new_image[pos_x + pos_y * new_width + new_size * r_g_b] = pixel(d_old_image, pos_x / scale, pos_y / scale, old_width, old_size, r_g_b) + (pos_y % scale) * ((pixel(d_old_image, pos_x / scale, pos_y / scale + 1, old_width, old_size, r_g_b) - pixel(d_old_image, pos_x / scale, pos_y / scale, old_width, old_size, r_g_b)) / (scale));
        else if (pos_y % scale == 0) d_new_image[pos_x + pos_y * new_width + new_size * r_g_b] = pixel(d_old_image, pos_x / scale, pos_y / scale, old_width, old_size, r_g_b) + (pos_x % scale) * ((pixel(d_old_image, pos_x / scale + 1, pos_y / scale, old_width, old_size, r_g_b) - pixel(d_old_image, pos_x / scale, pos_y / scale, old_width, old_size, r_g_b)) / (scale));
        else{
            int x_y_r = pixel(d_old_image, pos_x / scale, pos_y / scale, old_width, old_size, r_g_b) + (pos_x % scale) * ((pixel(d_old_image, pos_x / scale + 1, pos_y / scale, old_width, old_size, r_g_b) - pixel(d_old_image, pos_x / scale, pos_y / scale, old_width, old_size, r_g_b)) / (scale));
            int x_y_1_r = pixel(d_old_image, pos_x / scale, pos_y / scale + 1, old_width, old_size, r_g_b) + (pos_x % scale) * ((pixel(d_old_image, pos_x / scale + 1, pos_y / scale + 1, old_width, old_size, r_g_b) - pixel(d_old_image, pos_x / scale, pos_y / scale + 1, old_width, old_size, r_g_b)) / (scale));
            d_new_image[pos_x + pos_y * new_width + new_size * r_g_b] = x_y_r + (pos_y % scale) * ((x_y_1_r - x_y_r) / scale);
        }
    }
}

__global__ void nearest_neighbor_interpolation(unsigned char *d_old_image, unsigned char *d_new_image, int old_width, int old_height, int new_width, int new_height, int pixel_per_thread){
    int old_size = old_height * old_width;
    int new_size = new_height * new_width;
    float scale = (float)new_width / old_width;
    for (int i = 0; i < pixel_per_thread; i++){
        int pos = pixel_per_thread *(blockIdx.x * blockDim.x + threadIdx.x) + i;
        int pos_x = (pos % new_size) % new_width;
        int pos_y = (pos % new_size) / new_width;
        int r_g_b = pos / new_size;
        if (pos_x >= new_width || pos_y >= new_height) continue;
        d_new_image[pos_x + pos_y * new_width + new_size * r_g_b] = d_old_image[(int)(pos_x / scale) + (int)(pos_y / scale) * old_width + old_size * r_g_b];
    }
}

void interpolate(string path, string file_name, int scale, int interpolation_mode){
    CImg<unsigned char> img_in(path.c_str());
    int old_size = img_in.size() / 3;
    int old_width = img_in.width();
    int old_height = img_in.height();
    unsigned long long size = img_in.size();
    unsigned char *old_image = img_in.data();

    CImg<unsigned char> img_out;
    if(interpolation_mode == 1) img_out = CImg<unsigned char>(old_width * scale, old_height * scale, 1, 3, 255);
    if(interpolation_mode == 2) img_out = CImg<unsigned char>(old_width * scale - (scale - 1), old_height * scale - (scale - 1), 1, 3, 255);
    unsigned char *new_image = img_out.data();
    int new_size = img_out.size() / 3;
    int new_width = img_out.width();
    int new_height = img_out.height();

    unsigned char *d_old_image;
    unsigned char *d_new_image;

    hipMalloc((void **)&d_old_image, old_size * 3 * sizeof(unsigned char));
    hipMalloc((void **)&d_new_image, new_size * 3 * sizeof(unsigned char));

    TIMERSTART(parallel);
    hipMemcpy(d_old_image, old_image, old_size * 3, hipMemcpyHostToDevice);

    dim3 blkDim (block_size, 1, 1);
    dim3 grdDim ((((new_size * 3) + block_size - 1)/block_size + pixel_per_thread - 1)/pixel_per_thread, 1, 1);

    if(interpolation_mode == 1) nearest_neighbor_interpolation<<<grdDim, blkDim>>>(d_old_image, d_new_image, old_width, old_height, new_width, new_height, pixel_per_thread);
    if(interpolation_mode == 2) linear_interpolation<<<grdDim, blkDim>>>(d_old_image, d_new_image, old_width, old_height, new_width, new_height, scale, pixel_per_thread);

    hipDeviceSynchronize();

    hipMemcpy(new_image, d_new_image, new_size * 3, hipMemcpyDeviceToHost);

    TIMERSTOP(parallel);

    string _ = "new/";
    _.append(file_name);
    cout << "saving " << file_name << "..." << endl;
    img_out.save(_.c_str());
    hipFree(d_old_image);
    hipFree(new_image);
}

int main(int argc, char const *argv[]){
    int test = 0;
    int interpolation_mode = 0;
    int scale = 0;
    string path;
    if (argc < 4){
        cout << "Modo de uso: " << argv[0] << " \"Nombre imagen\" \"tecnica(NNI/LI)\" \"factor de escalado(ej: int >= 1)\"" << endl;
        return 1;
    }
    if(strcmp(argv[2], "NNI") == 0) interpolation_mode = 1;
    else if(strcmp(argv[2], "LI") == 0) interpolation_mode = 2;
    else{
        cout << "la tecnica de interpolacion puede ser \"NNI\" o \"LI\""<<endl;
        return 1;
    }
    scale = atoi(argv[3]);
    if(scale < 1){
        cout << "El factor de escalado debe ser un entero mayor o igual a 1" << endl;
        return 1;
    }
    if(argc > 4){
        if(strcmp(argv[4], "-t") == 0){
            cout <<"------------------ Test -------------------" << endl;
            test = 1;
        }
    }
    path = argv[1];

    if (auto dir = opendir(path.c_str())) { // leer todos los archivos de una carpeta
        while (auto f = readdir(dir)) {
            if (!f->d_name || f->d_name[0] == '.') continue; // Skip everything that starts with a dot
            printf("Processing %s...\n", f->d_name);
            string _ = argv[1];
            if(_.at(_.length() - 1) != '/') _.append("/");
            _.append(f->d_name);
            interpolate(_, f->d_name, scale, interpolation_mode);
        }
        closedir(dir);
    }
    return 0;
}